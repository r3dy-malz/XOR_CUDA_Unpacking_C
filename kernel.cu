#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

char* readFile(const char* file_path, long* file_size) {
    FILE* file = fopen(file_path, "r");
    if (file == NULL) {
        perror("Error opening file");
        return NULL;
    }

    fseek(file, 0, SEEK_END);
    *file_size = ftell(file);
    fseek(file, 0, SEEK_SET);

    char* file_data = (char*)malloc(*file_size + 1);
    if (file_data == NULL) {
        perror("Error allocating memory");
        fclose(file);
        return NULL;
    }
    fread(file_data, 1, *file_size, file);
    file_data[*file_size] = '\0';

    fclose(file);

    return file_data;
}

__global__ void xor (char* file_data_xored, char* file_data) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    file_data_xored[x] = file_data[x] ^ 0x43;
}

hipError_t xor_with_cuda(char* file_data_xored, char* file_data, long file_size) {
    hipError_t cudaStatus;
    char* dev_file_data;
    char* dev_file_data_xored;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_file_data, file_size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_file_data_xored, file_size * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_file_data, file_data, file_size * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    int blockSize = 256;
    int numBlocks = (file_size + blockSize - 1) / blockSize;
    xor << <numBlocks, blockSize >> > (dev_file_data_xored, dev_file_data);

    cudaStatus = hipMemcpy(file_data_xored, dev_file_data_xored, file_size * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    printf("test");
    hipFree(dev_file_data);
    hipFree(dev_file_data_xored);

    return cudaStatus;
}

void xor_with_cpu(char* file_data, long file_size) {
    if (file_data != NULL) {
        for (int x = 0; x < file_size; x++) {
            file_data[x] = file_data[x] ^ 0x43;
        }
    }
}

int main() {
    const char* file_path = "test_file.txt"; // Change THIS | GPU > CPU : File > 40 Mo
    long file_size;
    char* file_data = readFile(file_path, &file_size);

    char* file_data_xored = (char*)malloc(file_size + 1);

    //  CPU
    auto start_time_cpu = std::chrono::high_resolution_clock::now();

    xor_with_cpu(file_data, file_size); ///

    auto end_time_cpu = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> elapsed_time_cpu = end_time_cpu - start_time_cpu;
    printf("Temps d'exécution CPU : %f millisecondes\n", elapsed_time_cpu.count());
    printf("file_data:\n%s\n", file_data);
    //  GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    xor_with_cuda(file_data_xored, file_data, file_size); ///

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsed_time_gpu;
    hipEventElapsedTime(&elapsed_time_gpu, start, stop);
    printf("Temps d'exécution GPU : %f millisecondes\n", elapsed_time_gpu);
    printf("file_data:\n%s\n", file_data_xored);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    free(file_data);
    free(file_data_xored);

    return 0;
}